#include "hip/hip_runtime.h"
#include <algorithm> 
#include <iostream>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "CTCBeamSearch.h"

using namespace std;

// Utility

// Hash code C strings (same as Java hashcode())
int strHashCode(char* str, int len) {
    int hash = 0;
    if (len == 0) return hash;
    for (int i = 0; i < len; i++) {
        char c = str[i];
        hash = (31 * hash) + c;
    }
    return hash;
}

__device__ inline void genHashCode(char* str, int len, int* dest) {
    int hash = 0;
    if (len == 0) *dest = hash;
    for (int i = 0; i < len; i++) {
        char c = str[i];
        hash = (31 * hash) + c;
    }
    *dest = hash;
}

float* getRowData(float* data, int row, int length){
    float* ret= new float[length];
    memcpy(ret, data+row*length, length*sizeof(float));
    return ret;
}

// seq_len, 
float* getBatchAtT(float* devData, int timeIdx, int batchSize, int length) {
    return devData + timeIdx * batchSize * length;
}

void printMap (map<string, float> dict){
    for(map<string, float >::const_iterator it = dict.begin(); it != dict.end(); ++it)
    {
        std::cout << it->first << "," << it->second << ";";
    }

    std::cout<<endl;
}

void printSet (set<string> myset){
    std::set<std::string>::iterator it = myset.begin();
    while (it != myset.end())
    {
        std::cout << (*it) << ",";
        it++;
    }
    std::cout<<endl;
}

void printVector (vector<float> vec){
    std::vector<float>::iterator it = vec.begin();
    while (it != vec.end())
    {
        std::cout << (*it) << ",";
        it++;
    }
    std::cout<<endl;
}

void CTCBeamSearch::helper(){
    std::cout << "======print path======" << std::endl;
    printSet(path);
    std::cout << "======print pathScore======" << std::endl;
    printMap(pathScore);
}

// Setup

struct GlobalConstants {
    int vocabSize;
    int beamWidth;
    int blankID;
    // int decodeMaxLen;
    int batchSize;
    char* vocab;
};

__constant__ GlobalConstants cuConstParams;
// __device__ int cuNumPaths;

__device__ int my_mod_start = 0;
__device__ int my_mod(){
    return (my_mod_start++)/8;
}

__global__ void kernelGenerateSegmentAndIndex(int* segment, int* index, int size, int stride) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size) return;
    int value = tid / stride;
    segment[tid] = value;
    index[tid] = tid;
}

void CTCBeamSearch::batchSortByProb(float* batchProb, BeamState** beamStates, int* sortIdx, int* sortSegment, int batchSize) {
    int blockDim = 256;
    int numBlocks = (batchSize * beamWidth * vocabSize + blockDim - 1) / blockDim;
    kernelGenerateSegmentAndIndex<<<numBlocks, blockDim>>>(sortSegment, sortIdx, batchSize * beamWidth * vocabSize, beamWidth * vocabSize);
    int totalSize = batchSize * beamWidth * vocabSize;
    thrust::stable_sort_by_key(thrust::device, batchProb, batchProb + totalSize, sortIdx, thrust::greater<float>());
    // thrust::gather (need a different gather destination array)
}

void CTCBeamSearch::setup(int batchSize) {
    // move constants to GPU
    GlobalConstants params;
    params.vocabSize = vocabSize;
    params.beamWidth = beamWidth;
    params.blankID = blankID;
    params.batchSize = batchSize;
    // params.decodeMaxLen = DECODE_MAX_LEN;
    hipMalloc(&(params.vocab), vocabSize * sizeof(char));
    hipMemcpy(params.vocab, this->vocab, vocabSize * sizeof(char), hipMemcpyHostToDevice);
   
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

    // allocate buffers
    hipMalloc(&sortIdx, batchSize * beamWidth * vocabSize * sizeof(int));
    hipMalloc(&sortSegment, batchSize * beamWidth * vocabSize * sizeof(int));
    // int blockDim = 256;
    // int numBlocks = (batchSize * beamWidth * vocabSize + blockDim - 1) / blockDim;
    // kernelGenerateSegmentAndIndex<<<numBlocks, blockDim>>>(sortSegment, sortIdx, batchSize * beamWidth * vocabSize, beamWidth * vocabSize);
    
    hipError_t error;
    error = hipMalloc(&beamStates, batchSize * beamWidth * vocabSize * sizeof(BeamState*));
    error = hipMalloc(&nextBeamStates, batchSize * beamWidth * vocabSize * sizeof(BeamState*));
    error = hipMalloc(&beamStateBuffer, batchSize * beamWidth * vocabSize * sizeof(BeamState));
    error = hipMalloc(&nextBeamStateBuffer, batchSize * beamWidth * vocabSize * sizeof(BeamState));
    error = hipMalloc(&pathHashes, batchSize * beamWidth * vocabSize * sizeof(int));
    error = hipMalloc(&differentPathTest, batchSize * beamWidth * vocabSize * sizeof(int));
    error = hipMalloc(&mergedProbs, batchSize * beamWidth * vocabSize * sizeof(float));
    error = hipMalloc(&batchNumPaths, batchSize * sizeof(int));
    if (error != hipSuccess) {
        fprintf(stderr,"hipError_t: %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    }
}

// Decode

string CTCBeamSearch::decode(cuMatrix<float>* seqProb, int timestep, int batchSize) {
    setup(batchSize);
    // get time step
    // int timestep = seqProb->getRows();
    // check vocab size
    if(seqProb->getCols() != vocabSize){
        printf("Error: inconsistent vocabulary size in CTC decoder");
        exit(0);
    }

    // initial path at time t = 1
    float* initRow = seqProb->getDev();

    initialPath(getBatchAtT(seqProb->getDev(), 0, batchSize, vocabSize), batchSize);

    // iterate through timestep
    for (int t = 1; t < timestep; t++){
        float* prob = getBatchAtT(seqProb->getDev(), t, batchSize, vocabSize);
        extendAndPrune(prob, t == timestep - 1);
    }

    int bestLen;
    char best[DECODE_MAX_LEN];
    BeamState* bestState;
    hipMemcpy(&bestState, beamStates, sizeof(BeamState*), hipMemcpyDeviceToHost);
    hipMemcpy(&bestLen, &(bestState->len), sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(best, bestState->path, bestLen, hipMemcpyDeviceToHost);
    string best_string = string(best, bestLen);
    float bestScore;
    hipMemcpy(&bestScore, &(bestState->prob), sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Best Score: " << bestScore << std::endl; 
    return best_string;
}

// prob: batchSize, vocabSize
// beamStates, ...: batchSize, beamWidth * vocabSize
// batchNumPaths: batchSize,
__global__ void kernelInitialPath(float* prob, BeamState** beamStates, BeamState* beamStateBuffer, float* mergedProbs,
    int* batchNumPaths) {
    
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    int batchSize = cuConstParams.batchSize;
    int vocabSize = cuConstParams.vocabSize;
    if (pid >= vocabSize * batchSize) return;

    int beamWidth = cuConstParams.beamWidth;
    int blankID = cuConstParams.blankID;
    char* vocab = cuConstParams.vocab;

    int exampleIdx = pid / vocabSize; // example index inside each batch 
    int pathIdx = pid % vocabSize; // initialize vocabSize paths for each example in batch

    BeamState** exampleBeamStates = beamStates + exampleIdx * (beamWidth * vocabSize);
    BeamState* exampleBeamStateBuffer = beamStateBuffer + exampleIdx * (beamWidth * vocabSize);
    float* exampleMergedProbs = mergedProbs + exampleIdx * (beamWidth * vocabSize);

    exampleBeamStateBuffer[pathIdx].path[0] = vocab[pathIdx];
    float currProb = prob[exampleIdx * vocabSize + pathIdx];
    exampleBeamStateBuffer[pathIdx].prob = currProb;
    exampleBeamStateBuffer[pathIdx].len = 1;
    exampleBeamStates[pathIdx] = exampleBeamStateBuffer + pathIdx;
    
    exampleMergedProbs[pathIdx] = currProb;
    batchNumPaths[exampleIdx] = vocabSize;
}

void CTCBeamSearch::initialPath(float* prob, int batchSize) {
    // int s = 1;
    // BeamState* initialStates[vocabSize];
    // hipError_t error;
    // for (int i = 0; i < vocabSize; i++){
    //     error = hipMemset(&(beamStateBuffer[i].path), this->vocab[i], sizeof(char));
    //     error = hipMemcpy(&(beamStateBuffer[i].prob), &(prob[i]), sizeof(float), hipMemcpyDeviceToDevice);
    //     error = hipMemcpy(&(beamStateBuffer[i].len), &s, sizeof(int), hipMemcpyHostToDevice);
    //     initialStates[i] = beamStateBuffer + i;
    // }
    // error = hipMemcpy(beamStates, &initialStates, vocabSize * sizeof(BeamState*), hipMemcpyHostToDevice);
    
    // numPaths = vocabSize;
    // // error = hipMemcpyToSymbol(HIP_SYMBOL(cuNumPaths), &numPaths, sizeof(int));

    // // prune
    // error = hipMemcpy(mergedProbs, prob, vocabSize * sizeof(float), hipMemcpyDeviceToDevice);
    

    int blockDim = 256;
    int numBlocks = (batchSize * vocabSize + blockDim - 1) / blockDim;
    kernelInitialPath<<<numBlocks, blockDim>>>(prob, beamStates, beamStateBuffer, mergedProbs, batchNumPaths);
    
    numPaths = vocabSize;
    thrust::sort_by_key(thrust::device, mergedProbs, mergedProbs + numPaths, beamStates, thrust::greater<float>());
    numPaths = beamWidth > vocabSize ? vocabSize : beamWidth;

    // if (error != hipSuccess) {
    //     fprintf(stderr,"hipError_t: %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    // }
}

// TODO: handle length exceeding max len
__global__ void kernelGenNextPaths(float* vocabProbs, BeamState** beamStates, 
    BeamState** nextBeamStates, BeamState* beamStateBuffer, BeamState* nextBeamStateBuffer, 
    int* pathHashes, int numPaths, bool isLastStep) {

    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    int vocabSize = cuConstParams.vocabSize;
    // int beamWidth = cuConstParams.beamWidth;
    int blankID = cuConstParams.blankID;
    char* vocab = cuConstParams.vocab;

    if (pid >= vocabSize * numPaths) return;
    
    int pi = pid / vocabSize;
    int vi = pid % vocabSize;

    BeamState* newBeamState = &(nextBeamStateBuffer[pid]);
    BeamState* oldBeamState = beamStates[pi];
    nextBeamStates[pid] = newBeamState;
    char* newPath = newBeamState->path;
    char* oldPath = oldBeamState->path;
    memcpy(newPath, oldPath, DECODE_MAX_LEN * sizeof(char));
    newBeamState->prob = oldBeamState->prob * vocabProbs[vi];
    // extend with blank
    if (vi == blankID) {
        // path last char is blank
        if (oldPath[oldBeamState->len-1] == vocab[blankID]) {
            newBeamState->len = oldBeamState->len;
        } else {
            newBeamState->len = oldBeamState->len + 1;
            newPath[newBeamState->len-1] = vocab[vi]; // append new blank
        }
    } else {
        if (oldPath[oldBeamState->len-1] == vocab[blankID]) {
            newBeamState->len = oldBeamState->len;
            newPath[newBeamState->len-1] = vocab[vi]; // replace last blank with new char
        } else {
            if (oldPath[oldBeamState->len-1] == vocab[vi]) {
                newBeamState->len = oldBeamState->len;
            } else {
                newBeamState->len = oldBeamState->len + 1;
                newPath[newBeamState->len-1] = vocab[vi]; // append new char
            }
        }
    }
    if (isLastStep) {
        if (newPath[newBeamState->len-1] == vocab[blankID]) {
            newBeamState->len -= 1;
        }
    }
    genHashCode(newPath, newBeamState->len, &(pathHashes[pid]));
}

__global__ void kernelTestDifferentPaths(int* pathHashes, int* differentPathTest, int numPaths) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= numPaths) return;
    if (pid == 0 || pathHashes[pid] != pathHashes[pid-1]) differentPathTest[pid] = 1; 
}

__global__ void kernelkernel(float* a1, BeamState** a2) {
    // for cuda-gdb
}

__global__ void kernelMergeSamePaths(int* differentPathTest, BeamState** dest, BeamState** src, float* mergedProbs, int numPaths) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= numPaths) return;
    int dstIdx = differentPathTest[pid] - 1;
    dest[dstIdx] = src[pid]; 
    atomicAdd(mergedProbs + dstIdx, src[pid]->prob);
}

__global__ void kernelWriteMergedProbs(float* mergedProbs, BeamState** beamStates, int numPaths) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= numPaths) return;
    beamStates[pid]->prob = mergedProbs[pid];
}

void CTCBeamSearch::extendAndPrune(float* vocabProbs, bool isLastStep){
    // Assume: hipMalloc initialize memory to zero
    hipError_t error;
    error = hipMemset(mergedProbs, 0, vocabSize * beamWidth * sizeof(float));
    // generate all possible new paths (numPaths * vocabSize)
    int blockDim = 256;
    int numBlocks = (numPaths * vocabSize + blockDim - 1) / blockDim;
    kernelGenNextPaths<<<numBlocks, blockDim>>>(vocabProbs, beamStates, 
        nextBeamStates, beamStateBuffer, nextBeamStateBuffer, pathHashes, numPaths, isLastStep);
    
    // hipDeviceSynchronize();

    // sort by hash to group identical paths
    numPaths = numPaths * vocabSize;
    thrust::sort_by_key(thrust::device, pathHashes, pathHashes + numPaths, nextBeamStates);
    // test + scan to get index in merged array (unique paths)
    numBlocks = (numPaths * vocabSize + blockDim - 1) / blockDim;
    kernelTestDifferentPaths<<<numBlocks, blockDim>>>(pathHashes, differentPathTest, numPaths);
    thrust::inclusive_scan(thrust::device, differentPathTest, differentPathTest + numPaths, differentPathTest);
    // merge the probabilities of identical paths
    error = hipMemset(beamStates, 0, vocabSize * beamWidth * sizeof(BeamState*));
    kernelMergeSamePaths<<<numBlocks, blockDim>>>(differentPathTest, beamStates, nextBeamStates, mergedProbs, numPaths);
    // sort by probability
    error = hipMemcpy(&numPaths, (void *) (differentPathTest + numPaths - 1), sizeof(int), hipMemcpyDeviceToHost);
    thrust::sort_by_key(thrust::device, mergedProbs, mergedProbs + numPaths, beamStates, thrust::greater<float>());
    // prune
    numPaths = beamWidth > numPaths ? numPaths : beamWidth;
    // write merged probablities back to BeamState
    numBlocks = (numPaths * vocabSize + blockDim - 1) / blockDim;
    kernelWriteMergedProbs<<<numBlocks, blockDim>>>(mergedProbs, beamStates, numPaths);

    // std::swap(beamStates, nextBeamStates);
    std::swap(beamStateBuffer, nextBeamStateBuffer);

    error = hipMemset(nextBeamStateBuffer, 0, vocabSize * beamWidth * sizeof(BeamState));
    error = hipMemset(nextBeamStates, 0, vocabSize * beamWidth * sizeof(BeamState*));
    error = hipMemset(pathHashes, 0, vocabSize * beamWidth * sizeof(int));
    error = hipMemset(differentPathTest, 0, vocabSize * beamWidth * sizeof(int));
    if (error != hipSuccess) {
        fprintf(stderr,"hipError_t: %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    }
}