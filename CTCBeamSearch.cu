#include "hip/hip_runtime.h"
#include <algorithm> 
#include <iostream>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "CTCBeamSearch.h"

using namespace std;

// Utility

// Hash code C strings (same as Java hashcode())
int strHashCode(char* str, int len) {
    int hash = 0;
    if (len == 0) return hash;
    for (int i = 0; i < len; i++) {
        char c = str[i];
        hash = (31 * hash) + c;
    }
    return hash;
}

__device__ inline void genHashCode(char* str, int len, int* dest) {
    int hash = 0;
    if (len == 0) *dest = hash;
    for (int i = 0; i < len; i++) {
        char c = str[i];
        hash = (31 * hash) + c;
    }
    *dest = hash;
}

float* getRowData(float* data, int row, int length){
    float* ret= new float[length];
    memcpy(ret, data+row*length, length*sizeof(float));
    return ret;
}

float* getRowDataDev(float* devData, int row, int length) {
    return devData + row * length;
}

void printMap (map<string, float> dict){
    for(map<string, float >::const_iterator it = dict.begin(); it != dict.end(); ++it)
    {
        std::cout << it->first << "," << it->second << ";";
    }

    std::cout<<endl;
}

void printSet (set<string> myset){
    std::set<std::string>::iterator it = myset.begin();
    while (it != myset.end())
    {
        std::cout << (*it) << ",";
        it++;
    }
    std::cout<<endl;
}

void printVector (vector<float> vec){
    std::vector<float>::iterator it = vec.begin();
    while (it != vec.end())
    {
        std::cout << (*it) << ",";
        it++;
    }
    std::cout<<endl;
}

void CTCBeamSearch::helper(){
    std::cout << "======print path======" << std::endl;
    printSet(path);
    std::cout << "======print pathScore======" << std::endl;
    printMap(pathScore);
}

// Setup

struct GlobalConstants {
    int vocabSize;
    int beamWidth;
    int blankID;
    // int decodeMaxLen;
    char* vocab;
};

__constant__ GlobalConstants cuConstParams;
// __device__ int cuNumPaths;

void CTCBeamSearch::setup() {
    // move constants to GPU
    GlobalConstants params;
    params.vocabSize = vocabSize;
    params.beamWidth = beamWidth;
    params.blankID = blankID;
    // params.decodeMaxLen = DECODE_MAX_LEN;
    hipMalloc(&(params.vocab), vocabSize * sizeof(char));
    hipMemcpy(params.vocab, this->vocab, vocabSize * sizeof(char), hipMemcpyHostToDevice);
   
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

    // allocate buffers
    hipError_t error;
    error = hipMalloc(&beamStates, beamWidth * vocabSize * sizeof(BeamState*));
    error = hipMalloc(&nextBeamStates, beamWidth * vocabSize * sizeof(BeamState*));
    error = hipMalloc(&beamStateBuffer, beamWidth * vocabSize * sizeof(BeamState));
    error = hipMalloc(&nextBeamStateBuffer, beamWidth * vocabSize * sizeof(BeamState));
    error = hipMalloc(&pathHashes, beamWidth * vocabSize * sizeof(int));
    error = hipMalloc(&differentPathTest, beamWidth * vocabSize * sizeof(int));
    error = hipMalloc(&mergedProbs, beamWidth * vocabSize * sizeof(float));
    if (error != hipSuccess) {
        fprintf(stderr,"hipError_t: %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    }
}

// Decode

string CTCBeamSearch::decode(cuMatrix<float>* seqProb){
    setup();
    // get time step
    int timestep = seqProb->getRows();
    // check vocab size
    if(seqProb->getCols() != vocabSize){
        printf("Error: inconsistent vocabulary size in CTC decoder");
        exit(0);
    }

    // initial path at time t = 1
    float* initRow = seqProb->getDev();

    initialPath(getRowDataDev(seqProb->getDev(), 0, vocabSize));

    // iterate through timestep
    for (int t = 1; t < timestep; t++){
        float* prob = getRowDataDev(seqProb->getDev(), t, vocabSize);
        extendAndPrune(prob, t == timestep - 1);
    }

    char best[DECODE_MAX_LEN];
    BeamState* bestState;
    hipMemcpy(&bestState, beamStates, sizeof(BeamState*), hipMemcpyDeviceToHost);
    hipMemcpy(best, bestState->path, DECODE_MAX_LEN, hipMemcpyDeviceToHost);
    string best_string = best;
    return best_string;
}

void CTCBeamSearch::initialPath(float* prob){
    int s = 1;
    BeamState* initialStates[vocabSize];
    hipError_t error;
    for (int i = 0; i < vocabSize; i++){
        error = hipMemset(&(beamStateBuffer[i].path), this->vocab[i], sizeof(char));
        error = hipMemcpy(&(beamStateBuffer[i].prob), &(prob[i]), sizeof(float), hipMemcpyHostToDevice);
        error = hipMemcpy(&(beamStateBuffer[i].len), &s, sizeof(int), hipMemcpyHostToDevice);
        initialStates[i] = beamStateBuffer + i;
    }
    error = hipMemcpy(beamStates, &initialStates, vocabSize * sizeof(BeamState*), hipMemcpyHostToDevice);
    
    numPaths = vocabSize;
    // error = hipMemcpyToSymbol(HIP_SYMBOL(cuNumPaths), &numPaths, sizeof(int));

    // prune
    error = hipMemcpy(mergedProbs, prob, vocabSize * sizeof(float), hipMemcpyHostToDevice);
    thrust::sort_by_key(thrust::device, mergedProbs, mergedProbs + numPaths, beamStates, thrust::greater<float>());
    numPaths = beamWidth > vocabSize ? vocabSize : beamWidth;

    if (error != hipSuccess) {
        fprintf(stderr,"hipError_t: %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    }
}

// TODO: handle length exceeding max len
__global__ void kernelGenNextPaths(float* vocabProbs, BeamState** beamStates, 
    BeamState** nextBeamStates, BeamState* beamStateBuffer, BeamState* nextBeamStateBuffer, 
    int* pathHashes, int numPaths, bool isLastStep) {

    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    int vocabSize = cuConstParams.vocabSize;
    int beamWidth = cuConstParams.beamWidth;
    int blankID = cuConstParams.blankID;
    char* vocab = cuConstParams.vocab;

    if (pid >= vocabSize * numPaths) return;
    
    int pi = pid / vocabSize;
    int vi = pid % vocabSize;

    BeamState* newBeamState = &(nextBeamStateBuffer[pid]);
    BeamState* oldBeamState = beamStates[pi];
    nextBeamStates[pid] = newBeamState;
    char* newPath = newBeamState->path;
    char* oldPath = oldBeamState->path;
    memcpy(newPath, oldPath, DECODE_MAX_LEN * sizeof(char));
    newBeamState->prob = oldBeamState->prob * vocabProbs[vi];
    // extend with blank
    if (vi == blankID) {
        // path last char is blank
        if (oldPath[oldBeamState->len-1] == vocab[blankID]) {
            newBeamState->len = oldBeamState->len;
        } else {
            newBeamState->len = oldBeamState->len + 1;
            newPath[newBeamState->len-1] = vocab[vi]; // append new blank
        }
    } else {
        if (oldPath[oldBeamState->len-1] == vocab[blankID]) {
            newBeamState->len = oldBeamState->len;
            newPath[newBeamState->len-1] = vocab[vi]; // replace last blank with new char
        } else {
            if (oldPath[oldBeamState->len-1] == vocab[vi]) {
                newBeamState->len = oldBeamState->len;
            } else {
                newBeamState->len = oldBeamState->len + 1;
                newPath[newBeamState->len-1] = vocab[vi]; // append new char
            }
        }
    }
    if (isLastStep) {
        if (newPath[newBeamState->len-1] == vocab[blankID]) {
            newBeamState->len -= 1;
        }
    }
    genHashCode(newPath, newBeamState->len, &(pathHashes[pid]));
}

__global__ void kernelTestDifferentPaths(int* pathHashes, int* differentPathTest, int numPaths) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= numPaths) return;
    if (pid == 0 || pathHashes[pid] != pathHashes[pid-1]) differentPathTest[pid] = 1; 
}

__global__ void kernelkernel(float* a1, BeamState** a2) {
    // for cuda-gdb
}

__global__ void kernelMergeSamePaths(int* differentPathTest, BeamState** dest, BeamState** src, float* mergedProbs, int numPaths) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= numPaths) return;
    int dstIdx = differentPathTest[pid] - 1;
    dest[dstIdx] = src[pid]; 
    atomicAdd(mergedProbs + dstIdx, src[pid]->prob);
}

__global__ void kernelWriteMergedProbs(float* mergedProbs, BeamState** beamStates, int numPaths) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= numPaths) return;
    beamStates[pid]->prob = mergedProbs[pid];
}

void CTCBeamSearch::extendAndPrune(float* vocabProbs, bool isLastStep){
    // Assume: hipMalloc initialize memory to zero
    hipError_t error;
    error = hipMemset(mergedProbs, 0, vocabSize * beamWidth * sizeof(float));
    // generate all possible new paths (numPaths * vocabSize)
    int blockDim = 256;
    int numBlocks = (numPaths * vocabSize + blockDim - 1) / blockDim;
    kernelGenNextPaths<<<numBlocks, blockDim>>>(vocabProbs, beamStates, 
        nextBeamStates, beamStateBuffer, nextBeamStateBuffer, pathHashes, numPaths, isLastStep);
    
    // hipDeviceSynchronize();

    // sort by hash to group identical paths
    numPaths = numPaths * vocabSize;
    thrust::sort_by_key(thrust::device, pathHashes, pathHashes + numPaths, nextBeamStates);
    // test + scan to get index in merged array (unique paths)
    numBlocks = (numPaths * vocabSize + blockDim - 1) / blockDim;
    kernelTestDifferentPaths<<<numBlocks, blockDim>>>(pathHashes, differentPathTest, numPaths);
    thrust::inclusive_scan(thrust::device, differentPathTest, differentPathTest + numPaths, differentPathTest);
    // merge the probabilities of identical paths
    error = hipMemset(beamStates, 0, vocabSize * beamWidth * sizeof(BeamState*));
    kernelMergeSamePaths<<<numBlocks, blockDim>>>(differentPathTest, beamStates, nextBeamStates, mergedProbs, numPaths);
    // sort by probability
    error = hipMemcpy(&numPaths, (void *) (differentPathTest + numPaths - 1), sizeof(int), hipMemcpyDeviceToHost);
    thrust::sort_by_key(thrust::device, mergedProbs, mergedProbs + numPaths, beamStates, thrust::greater<float>());
    // prune
    numPaths = beamWidth > numPaths ? numPaths : beamWidth;
    // write merged probablities back to BeamState
    numBlocks = (numPaths * vocabSize + blockDim - 1) / blockDim;
    kernelWriteMergedProbs<<<numBlocks, blockDim>>>(mergedProbs, beamStates, numPaths);

    // std::swap(beamStates, nextBeamStates);
    std::swap(beamStateBuffer, nextBeamStateBuffer);

    error = hipMemset(nextBeamStateBuffer, 0, vocabSize * beamWidth * sizeof(BeamState));
    error = hipMemset(nextBeamStates, 0, vocabSize * beamWidth * sizeof(BeamState*));
    error = hipMemset(pathHashes, 0, vocabSize * beamWidth * sizeof(int));
    error = hipMemset(differentPathTest, 0, vocabSize * beamWidth * sizeof(int));
    if (error != hipSuccess) {
        fprintf(stderr,"hipError_t: %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
    }
}